#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>
#include <tuple>

using c10::cuda::CUDAStreamGuard;
using c10::cuda::getCurrentCUDAStream;
using std::make_tuple;
using torch::autograd::AutogradContext;
using torch::autograd::Function;
using torch::autograd::tensor_list;
using torch::PackedTensorAccessor32;
using torch::RestrictPtrTraits;
using torch::Tensor;
using torch::TensorOptions;

template <typename scalar_t, int num_dims>
    using Accessor = PackedTensorAccessor32<scalar_t, num_dims, RestrictPtrTraits>;

template <typename scalar_t, int num_dims> 
inline Accessor<scalar_t, num_dims> get_accessor(const Tensor& tensor) {
    return tensor.packed_accessor32<scalar_t, num_dims, RestrictPtrTraits>();
};

template <typename scalar_t> __device__ __forceinline__ scalar_t sqrt_(scalar_t x) {};
template<> __device__ __forceinline__ float sqrt_(float x) { return ::sqrtf(x); };
template<> __device__ __forceinline__ double sqrt_(double x) { return ::sqrt(x); };

template <typename scalar_t> __global__ void forward_kernel(
    const Accessor<scalar_t, 2> positions,
    Accessor<int32_t, 1> rows,
    Accessor<int32_t, 1> columns,
    Accessor<scalar_t, 2> deltas,
    Accessor<scalar_t, 1> distances
) {
    const int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t num_neighbors = distances.size(0);
    if (index >= num_neighbors) return;

    int32_t row = floor((sqrtf(8 * index + 1) + 1) / 2);
    if (row * (row - 1) > 2 * index) row--;
    const int32_t column = index - row * (row - 1) / 2;

    const scalar_t delta_x = positions[row][0] - positions[column][0];
    const scalar_t delta_y = positions[row][1] - positions[column][1];
    const scalar_t delta_z = positions[row][2] - positions[column][2];
    const scalar_t distance = sqrt_(delta_x * delta_x + delta_y * delta_y + delta_z * delta_z);

    rows[index] = row;
    columns[index] = column;
    deltas[index][0] = delta_x;
    deltas[index][1] = delta_y;
    deltas[index][2] = delta_z;
    distances[index] = distance;
}

template <typename scalar_t> __global__ void backward_kernel(
    const Accessor<int32_t, 1> rows,
    const Accessor<int32_t, 1> columns,
    const Accessor<scalar_t, 2> deltas,
    const Accessor<scalar_t, 1> distances,
    const Accessor<scalar_t, 1> grad_distances,
    Accessor<scalar_t, 2> grad_positions
) {
    const int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t num_neighbors = distances.size(0);
    if (index >= num_neighbors) return;

    const scalar_t grad = grad_distances[index] / distances[index];
    const scalar_t grad_x = deltas[index][0] * grad;
    const scalar_t grad_y = deltas[index][1] * grad;
    const scalar_t grad_z = deltas[index][2] * grad;

    const int32_t row = rows[index];
    atomicAdd(&grad_positions[row][0], grad_x);
    atomicAdd(&grad_positions[row][1], grad_y);
    atomicAdd(&grad_positions[row][2], grad_z);

    const int32_t column = columns[index];
    atomicAdd(&grad_positions[column][0], -grad_x);
    atomicAdd(&grad_positions[column][1], -grad_y);
    atomicAdd(&grad_positions[column][2], -grad_z);
}

class Autograd : public Function<Autograd> {
public:
    static tensor_list forward(AutogradContext* ctx, const Tensor positions) {

        TORCH_CHECK(positions.dim() == 2, "Expected \"positions\" to have two dimensions");
        TORCH_CHECK(positions.size(0) > 0, "Expected the 1nd dimension size of \"positions\" to be more than 0");
        TORCH_CHECK(positions.size(1) == 3, "Expected the 2nd dimension size of \"positions\" to be 3");
        TORCH_CHECK(positions.is_contiguous(), "Expected \"positions\" to be contiguous");

        const int num_atoms = positions.size(0);
        const int num_neighbors = num_atoms * (num_atoms - 1) / 2;
        const int num_threads = 128;
        const int num_blocks = (num_neighbors + num_threads - 1) / num_threads;
        const auto stream = getCurrentCUDAStream(positions.get_device());

        const TensorOptions options = positions.options();
        const Tensor rows = torch::empty(num_neighbors, options.dtype(torch::kInt32));
        const Tensor columns = torch::empty(num_neighbors, options.dtype(torch::kInt32));
        const Tensor deltas = torch::empty({num_neighbors, 3}, options);
        const Tensor distances = torch::empty(num_neighbors, options);

        AT_DISPATCH_FLOATING_TYPES(positions.scalar_type(), "get_neighbor_list", [&]() {
            const CUDAStreamGuard guard(stream);
            forward_kernel<<<num_blocks, num_threads, 0, stream>>>(
                get_accessor<scalar_t, 2>(positions),
                get_accessor<int32_t, 1>(rows),
                get_accessor<int32_t, 1>(columns),
                get_accessor<scalar_t, 2>(deltas),
                get_accessor<scalar_t, 1>(distances));
        });

        ctx->save_for_backward({rows, columns, deltas, distances});
        ctx->saved_data["num_atoms"] = num_atoms;

        return {rows, columns, distances};
    }

    static tensor_list backward(AutogradContext* ctx, tensor_list grad_inputs) {

        const Tensor grad_distances = grad_inputs[2];
        const int num_atoms = ctx->saved_data["num_atoms"].toInt();
        const int num_neighbors = grad_distances.size(0);
        const int num_threads = 128;
        const int num_blocks = (num_neighbors + num_threads - 1) / num_threads;
        const auto stream = getCurrentCUDAStream(grad_distances.get_device());

        const tensor_list neighbors = ctx->get_saved_variables();
        const Tensor rows = neighbors[0];
        const Tensor columns = neighbors[1];
        const Tensor deltas = neighbors[2];
        const Tensor distances = neighbors[3];
        const Tensor grad_positions = torch::zeros({num_atoms, 3}, grad_distances.options());

        AT_DISPATCH_FLOATING_TYPES(grad_distances.scalar_type(), "get_neighbor_list", [&]() {
            const CUDAStreamGuard guard(stream);
            backward_kernel<<<num_blocks, num_threads, 0, stream>>>(
                get_accessor<int32_t, 1>(rows),
                get_accessor<int32_t, 1>(columns),
                get_accessor<scalar_t, 2>(deltas),
                get_accessor<scalar_t, 1>(distances),
                get_accessor<scalar_t, 1>(grad_distances),
                get_accessor<scalar_t, 2>(grad_positions));
        });

        return {grad_positions};
      }
};

TORCH_LIBRARY_IMPL(neighbors, AutogradCUDA, m) {
    m.impl("get_neighbor_list", [](const Tensor& positions){
        const tensor_list neighbors = Autograd::apply(positions);
        return make_tuple(neighbors[0], neighbors[1], neighbors[2]);
    });
}